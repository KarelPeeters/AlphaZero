#include "hip/hip_runtime.h"
__host__ __device__ int ceil_div(int x, int y) {
    return (x + y - 1) / y;
}

__device__ int2 fast_div(int a, int b) {
    int q, r;

    // fast path for powers of two (including b=1)
    if (b & (b - 1) == 0) {
        q = a >> __popc(b - 1);
        r = a - q * b;
    } else {
        q = a / b;
        r = a % b;
    }

    return make_int2(q, r);
}

template<typename T, int R>
struct Array {
    T data[R];

    __device__ T &operator[](int index) {
        return this->data[index];
    }
};

// de-dollar-ify template parameters
const int SIZE = $SIZE$;
const int RANK = $RANK$;
const int OPERANDS = $OPERANDS$;
const int STRIDES_DENSE[RANK] = $STRIDES_DENSE$;
const int STRIDES[OPERANDS][RANK] = $STRIDES$;

__device__ void operation(void *pointers[OPERANDS], int offsets[OPERANDS]) {
    $OPERATION$;
}

__global__ void scalar_kernel(
        Array<void *, OPERANDS> pointers
) {
    // common startup constants
    const int blockCount = gridDim.x;
    const int threadsPerBlock = blockDim.x;
    const int threadCount = blockCount * threadsPerBlock;

    const int block = blockIdx.x;
    const int thread = threadIdx.x;
    const int global = block * threadsPerBlock + thread;

    const int itemsPerThread = ceil_div(SIZE, threadCount);

    // the main loop, following https://developer.nvidia.com/blog/cuda-pro-tip-write-flexible-kernels-grid-stride-loops/
    for (int flat = global; flat < SIZE; flat += threadCount) {
        // convert the flat index into a per-operand offset
        int flat_left = flat;
        int offsets[OPERANDS] = {};

        for (int axis = 0; axis < RANK; axis++) {
            int2 result = fast_div(flat_left, STRIDES_DENSE[axis]);
            int axis_index = result.x;
            flat_left = result.y;

            for (int operand = 0; operand < OPERANDS; operand++) {
                offsets[operand] += axis_index * STRIDES[operand][axis];
            }
        }

        // actually run the operation
        operation(pointers.data, &offsets[0]);
    }
}
