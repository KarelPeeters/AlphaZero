#include "hip/hip_runtime.h"
// Implementation based on FlashAttention
//   paper: https://arxiv.org/abs/2205.14135
//   repository: https://github.com/HazyResearch/flash-attention

#include "util.cu"

// TODO proper stride support
// TODO properly implement reduce operations
// TODO mask and scale

// de-dollar-ify template parameters
const int S = $S$;
const int D = $D$;

// block sizes
const int B_QO = $B_QO$;
const int B_KV = $B_KV$;

// TODO relax this requirement?
static_assert(true && (S % B_QO == 0) && S % B_KV == 0 && D % B_KV == 0, "block sizes must divide S and D");
const int BC_QO = ceil_div(S, B_QO);
const int BC_KV = ceil_div(S, B_KV);

const int SCRATCH_SIZE = $SCRATCH_SIZE$;
static_assert(true && SCRATCH_SIZE == 2 * S, "Scratch size mismatch");

__global__ void attention_kernel(
        float *global_q, float *global_k, float *global_v,
        float *global_o, float *scratch
) {
    KernelInfo info = kernel_info();
    float *global_max = scratch;
    float *global_sum = scratch + S;

    assert(info.threads_per_block == B_QO * B_KV);
    // TODO also try transposing this
    int thread_qo_i = info.thread_id / B_KV;
    int thread_kv_j = info.thread_id % B_KV;
    bool is_first_q_thread = thread_kv_j == 0;

    // zero-initialize output and scratch
    for (int i = info.thread_id; i < S * D; i += info.threads_per_block) {
        global_o[i] = 0.0;
    }
    for (int i = info.thread_id; i < S; i += info.threads_per_block) {
        global_max[i] = -1.0 / 0.0;
        global_sum[i] = 0.0;
    }

    __syncthreads();

    // local memory
    __shared__ float block_q[B_QO * D];
    __shared__ float block_k[B_KV * D];
    __shared__ float block_v[B_KV * D];
    __shared__ float block_o[B_QO * D];
    // TODO try fusing old/new to reduce shared mem usage
    __shared__ float block_max_old[B_QO];
    __shared__ float block_max_new[B_QO];
    __shared__ float block_sum_old[B_QO];
    __shared__ float block_sum_new[B_QO];
    __shared__ float block_logits[B_QO][B_KV];

    // main processing
    for (int block_kv_j = 0; block_kv_j < BC_KV; block_kv_j++) {
        /* Load inputs */
        // load k, v
        for (int i = info.thread_id; i < B_KV * D; i += info.threads_per_block) {
            int offset = block_kv_j * B_KV * D;
            block_k[i] = global_k[offset + i];
            block_v[i] = global_v[offset + i];
        }
        __syncthreads();

        for (int block_qo_i = 0; block_qo_i < BC_QO; block_qo_i++) {
            // load q, o
            for (int i = info.thread_id; i < B_QO * D; i += info.threads_per_block) {
                int offset = block_qo_i * B_QO * D;
                block_q[i] = global_q[offset + i];
                block_o[i] = global_o[offset + i];
            }

            // load max, sum
            if (is_first_q_thread) {
                int offset = block_qo_i * B_QO;
                block_max_old[thread_qo_i] = global_max[offset + thread_qo_i];
                block_sum_old[thread_qo_i] = global_sum[offset + thread_qo_i];
            }
            __syncthreads();

            /* Compute deltas */
            // compute logits, each thread does one row/col dot product
            {
                float curr_logit = 0.0;
                for (int d = 0; d < D; d++) {
                    curr_logit += block_q[thread_qo_i * D + d] * block_k[thread_kv_j * D + d];
                }
                block_logits[thread_qo_i][thread_kv_j] = curr_logit;
            }
            __syncthreads();

            // compute new max per query
            if (is_first_q_thread) {
                float curr_max = block_max_old[thread_qo_i];
                for (int j = 0; j < B_KV; j++) {
                    curr_max = max(curr_max, block_logits[thread_qo_i][j]);
                }
                block_max_new[thread_qo_i] = curr_max;
            }
            __syncthreads();

            // compute exp(logit-max)
            block_logits[thread_qo_i][thread_kv_j] = expf(
                    block_logits[thread_qo_i][thread_kv_j] - block_max_new[thread_qo_i]
            );
            __syncthreads();

            // compute new sum per query
            if (is_first_q_thread) {
                float scalar_old = expf(block_max_old[thread_qo_i] - block_max_new[thread_qo_i]);
                float curr_sum = scalar_old * block_sum_old[thread_qo_i];
                for (int j = 0; j < B_KV; j++) {
                    curr_sum += block_logits[thread_qo_i][j];
                }
                block_sum_new[thread_qo_i] = curr_sum;
            }
            __syncthreads();

            // compute output
            // every thread calculates (D / B_KV) output values
            static_assert(D % B_KV == 0, "B_KV must divide D");

            float scale_old = block_sum_old[thread_qo_i]
                              * expf(block_max_old[thread_qo_i] - block_max_new[thread_qo_i]);
            float scale_shared = 1.0f / block_sum_new[thread_qo_i];

            for (int d = thread_kv_j; d < D; d += B_KV) {
                float o_delta_curr = 0.0;
                for (int j = 0; j < B_KV; j++) {
                    o_delta_curr += block_logits[thread_qo_i][j] * block_v[j * D + d];
                }
                block_o[thread_qo_i * D + d] = scale_shared * (scale_old * block_o[thread_qo_i * D + d] + o_delta_curr);
            }
            __syncthreads();

            /* Store outputs */
            // store o
            for (int i = info.thread_id; i < B_QO * D; i += info.threads_per_block) {
                int offset = block_qo_i * B_QO * D;
                global_o[offset + i] = block_o[i];
            }

            // store max, sum
            if (is_first_q_thread) {
                int offset = block_qo_i * B_QO;
                global_max[offset + thread_qo_i] = block_max_new[thread_qo_i];
                global_sum[offset + thread_qo_i] = block_sum_new[thread_qo_i];
            }

            __syncthreads();
        }
    }
}

// OLD attempt
/*

for (int b_kv = 0; b_kv < T_kv; b_kv++) {
    // load Kj and Vj (with grid stride loop)
    for (int a = info.thread_id; a < B_kv * D; a += info.threads_per_block) {
        k_shared[a] = input_k[b_kv * B_kv * D + a];
        v_shared[a] = input_v[b_kv * B_kv * D + a];
    }

    __syncthreads();

    for (int b_qo = 0; b_qo < T_qo; b_qo++) {
        // load Qi and Oi (with grid stride loop)
        for (int a = info.thread_id; a < B_qo * D; a += info.threads_per_block) {
            q_shared[a] = input_q[b_qo * B_qo * D + a];
            o_shared[a] = output[b_qo * B_qo * D + a];
        }

        __syncthreads();

        // compute Sij = Qi * Kj.T, each thread computes one element
        __shared__ float logit_buffer[B_qo][B_kv];
        float logit_thread = 0.0;
        for (int d = 0; d < D; d++) {
            logit_thread += q_shared[thread_i * D + d] * k_shared[thread_j * D + d];
        }
        logit_buffer[b_qo][b_kv] = logit_thread;
        __syncthreads();

        // reduce max over keys
        __shared__ float max_buffer[B_qo];
        if (b_kv == 0) {
            float curr_max = -1.0 / 0.0;
            for (int i = 0; i < B_kv; i++) {
                curr_max = max(curr_max, logit_buffer[b_qo][i]);
            }
            max_buffer[b_qo] = curr_max;
        }
        __syncthreads();

        // map exponential
        logit_thread = expf(logit_thread - max_buffer[b_qo]);
        logit_buffer[b_qo][b_kv] = logit_thread;
        __syncthreads();

        // reduce sum over keys
        __shared__ float sum_buffer[B_qo];
        if (b_kv == 0) {
            float curr_sum = 0.0;
            for (int i = 0; i < B_kv; i++) {
                curr_sum += logit_buffer[b_qo][i];
            }
            sum_buffer[b_qo] = curr_sum;
        }
        __syncthreads();


        // TODO continue here
        // TODO maybe rewrite the pseudocode in Python and with some better var names
    }
}

 */