#include "hip/hip_runtime.h"
// Implementation based on FlashAttention
//   paper: https://arxiv.org/abs/2205.14135
//   repository: https://github.com/HazyResearch/flash-attention

#include "util.cu"

// TODO proper stride support
// TODO properly implement reduce operations
// TODO mask and scale
// TODO more general bank conflict fix, this is basically hardcoded to D being a multiple of 32

// de-dollar-ify template parameters
const int S = $S$;
const int D = $D$;

// block sizes
const int B_QO = $B_QO$;
const int B_KV = $B_KV$;

// TODO relax this requirement?
static_assert(true && (S % B_QO == 0) && S % B_KV == 0 && D % B_KV == 0, "block sizes must divide S and D");
const int BC_QO = ceil_div(S, B_QO);
const int BC_KV = ceil_div(S, B_KV);

const int SCRATCH_SIZE = $SCRATCH_SIZE$;
static_assert(true && SCRATCH_SIZE == 2 * S, "Scratch size mismatch");

// launch_bounds ensures the compiler doesn't use too many registers
__global__ void __launch_bounds__(B_QO * B_KV)

attention_kernel(
        float *global_q, float *global_k, float *global_v,
        float *global_o, float *scratch
) {
    KernelInfo info = kernel_info();
    float *global_max = scratch;
    float *global_sum = scratch + S;

    assert(info.threads_per_block == B_QO * B_KV);
    // TODO also try transposing this
    int thread_qo_i = info.thread_id / B_KV;
    int thread_kv_j = info.thread_id % B_KV;
    bool is_first_q_thread = thread_kv_j == 0;

    // zero-initialize output and scratch
    for (int i = info.thread_id; i < S * D; i += info.threads_per_block) {
        global_o[i] = 0.0;
    }
    for (int i = info.thread_id; i < S; i += info.threads_per_block) {
        global_max[i] = -1.0 / 0.0;
        global_sum[i] = 0.0;
    }

    __syncthreads();

    // local memory
    __shared__ float block_q[B_QO * D];
    __shared__ float block_k[B_KV * (D + 1)];
    __shared__ float block_v[B_KV * D];
    __shared__ float block_o[B_QO * D];
    // TODO try fusing old/new to reduce shared mem usage
    __shared__ float block_max_old[B_QO];
    __shared__ float block_max_new[B_QO];
    __shared__ float block_sum_old[B_QO];
    __shared__ float block_sum_new[B_QO];
    __shared__ float block_logits[B_QO][B_KV];

    // main processing
    for (int block_kv_j = 0; block_kv_j < BC_KV; block_kv_j++) {
        /* Load inputs */
        // load k, v
        for (int i = info.thread_id; i < B_KV * D; i += info.threads_per_block) {
            int offset = block_kv_j * B_KV * D;
            int pad_i = (i / D) * (D + 1) + (i % D);
            block_k[pad_i] = global_k[offset + i];
            block_v[i] = global_v[offset + i];
        }
        __syncthreads();

        for (int block_qo_i = 0; block_qo_i < BC_QO; block_qo_i++) {
            // load q, o
            for (int i = info.thread_id; i < B_QO * D; i += info.threads_per_block) {
                int offset = block_qo_i * B_QO * D;
                block_q[i] = global_q[offset + i];
                block_o[i] = global_o[offset + i];
            }

            // load max, sum
            if (is_first_q_thread) {
                int offset = block_qo_i * B_QO;
                block_max_old[thread_qo_i] = global_max[offset + thread_qo_i];
                block_sum_old[thread_qo_i] = global_sum[offset + thread_qo_i];
            }
            __syncthreads();

            /* Compute deltas */
            // compute logits, each thread does one row/col dot product
            {
                float curr_logit = 0.0;
                for (int d = 0; d < D; d++) {
                    float q_value = block_q[thread_qo_i * D + d]; // bank broadcast
                    float k_value = block_k[thread_kv_j * (D + 1) + d]; // avoid bank conflict
                    curr_logit += q_value * k_value;
                }
                block_logits[thread_qo_i][thread_kv_j] = curr_logit;
            }
            __syncthreads();

            // compute new max per query
            if (is_first_q_thread) {
                float curr_max = block_max_old[thread_qo_i];
                for (int j = 0; j < B_KV; j++) {
                    curr_max = max(curr_max, block_logits[thread_qo_i][j]);
                }
                block_max_new[thread_qo_i] = curr_max;
            }
            __syncthreads();

            // compute exp(logit-max)
            block_logits[thread_qo_i][thread_kv_j] = expf(
                    block_logits[thread_qo_i][thread_kv_j] - block_max_new[thread_qo_i]
            );
            __syncthreads();

            // compute new sum per query
            if (is_first_q_thread) {
                float scalar_old = expf(block_max_old[thread_qo_i] - block_max_new[thread_qo_i]);
                float curr_sum = scalar_old * block_sum_old[thread_qo_i];
                for (int j = 0; j < B_KV; j++) {
                    curr_sum += block_logits[thread_qo_i][j];
                }
                block_sum_new[thread_qo_i] = curr_sum;
            }
            __syncthreads();

            // compute output
            // every thread calculates (D / B_KV) output values
            static_assert(D % B_KV == 0, "B_KV must divide D");

            float scale_old = block_sum_old[thread_qo_i]
                              * expf(block_max_old[thread_qo_i] - block_max_new[thread_qo_i]);
            float scale_shared = 1.0f / block_sum_new[thread_qo_i];

            for (int d = thread_kv_j; d < D; d += B_KV) {
                float o_delta_curr = 0.0;
                for (int j = 0; j < B_KV; j++) {
                    o_delta_curr += block_logits[thread_qo_i][j] * block_v[j * D + d];
                }
                block_o[thread_qo_i * D + d] = scale_shared * (scale_old * block_o[thread_qo_i * D + d] + o_delta_curr);
            }
            __syncthreads();

            /* Store outputs */
            // store o
            for (int i = info.thread_id; i < B_QO * D; i += info.threads_per_block) {
                int offset = block_qo_i * B_QO * D;
                global_o[offset + i] = block_o[i];
            }

            // store max, sum
            if (is_first_q_thread) {
                int offset = block_qo_i * B_QO;
                global_max[offset + thread_qo_i] = block_max_new[thread_qo_i];
                global_sum[offset + thread_qo_i] = block_sum_new[thread_qo_i];
            }

            __syncthreads();
        }
    }
}
