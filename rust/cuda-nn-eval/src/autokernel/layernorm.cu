#include "hip/hip_runtime.h"
#include "util.cu"

// de-dollar-ify template parameters
const int RANK = $RANK$;
const int STATIC_SIZE = $STATIC_SIZE$;
const int NORM_SIZE = $NORM_SIZE$;

// *CAREFUL* these arrays are actually of length RANK-1, but zero-sized arrays are not allowed in C++ so we pad them
const int STATIC_DENSE_STRIDES[RANK] = $STATIC_DENSE_STRIDES$;
const int STATIC_STRIDES[2][RANK] = $STATIC_STRIDES$;

const int NORM_STRIDES[2] = $NORM_STRIDES$;

// Every block handles a single layernorm group.
// Uses Welford's algorithm to compute the mean and variance
//   (see https://en.wikipedia.org/wiki/Algorithms_for_calculating_variance#Welford's_online_algorithm).
__global__ void layernorm_kernel(
        float *input,
        float *output,
        float eps
) {
    KernelInfo info = kernel_info();

    int static_index = info.global_warp_id;
    if (static_index >= STATIC_SIZE) {
        return;
    }

    Array<int, 2> static_offsets = flat_index_to_offsets<RANK, 2>(static_index, STATIC_DENSE_STRIDES, STATIC_STRIDES);

    float cache[ceil_div(NORM_SIZE, 32)];

    int count = 0;
    float mean = 0.0;
    float m2 = 0.0;

    // fill cache and calculate max
    for (int i = info.lane_id; i < NORM_SIZE; i += 32) {
        int offset = static_offsets[0] + i * NORM_STRIDES[0];
        float curr_raw = input[offset];

        cache[i / 32] = curr_raw;

//        printf("Thread %d loaded %f\n", info.global_thread_id, curr_raw);

        count += 1;
        float delta = curr_raw - mean;
        mean += delta / count;
        m2 += delta * (curr_raw - mean);
    }
//    printf("Thread %d calculated count %d mean %f and variance %f\n", info.global_thread_id, count, mean, m2);

    // combine variance and mean between threads
    for (int offset = 16; offset > 0; offset /= 2) {
        int next_count = __shfl_down_sync(FULL_WARP_MASK, count, offset);
        float next_mean = __shfl_down_sync(FULL_WARP_MASK, mean, offset);
        float next_m2 = __shfl_down_sync(FULL_WARP_MASK, m2, offset);

//        printf("Thread %d combining (%d, %f, %f) with (%d, %f, %f)\n", info.global_thread_id, count, mean, m2, next_count, next_mean, next_m2);

        int prev_count = count;
        count += next_count;

        float delta = next_mean - mean;
        float factor = (float) next_count / (float) count;

        if (factor != factor) {
            factor = 0.0;
        }

        mean += delta * factor;
        m2 += next_m2 + delta * delta * prev_count * factor;

//        printf("Thread %d combined (%d, %f, %f)\n", info.global_thread_id, count, mean, m2);
    }

    float var = m2 / count;
    float denom = sqrt(var + eps);

    // broadcast to all threads
    mean = __shfl_sync(FULL_WARP_MASK, mean, 0);
    denom = __shfl_sync(FULL_WARP_MASK, denom, 0);

    // normalize and write to output
    for (int i = info.lane_id; i < NORM_SIZE; i += 32) {
        int offset = static_offsets[1] + i * NORM_STRIDES[1];
        float x = cache[i / 32];
        float y = (x - mean) / denom;
        output[offset] = y;
    }
}